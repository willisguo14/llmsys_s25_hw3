#include "hip/hip_runtime.h"
#include <math.h>

#include <cub/block/block_load.cuh>
#include <hipcub/hipcub.hpp>

#include "includes/block_reduce.h"
#include "includes/kernels.h"

#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;
const float EPSILON = 1e-8f;

namespace lightseq {
namespace cuda {
/**
@brief: softmax_kernel
Softmax forward kernel for
  enc-self-attn, dec-self-attn, encdec-attn

@thread
gridDim.x = dynamic
gridDim.y = batch_size
gridDim.z = nhead
blockDim.x = from_len

@param
inp: [batch_size, nhead, from_len, to_len], softmax input.
attn_mask: [batch_size, to_len], padding tokens are -inf,
  non padding tokens are 0.
  attn_mask!=nullptr for enc-self-attn and enc-dec-attn
  attn_mask=nullptr and mask_future=ture for dec-self-attn training
  attn_mask=nullptr and mask_future=false for dec-self-attn infer
*/
template <typename T, int block_dim, int ele_per_thread>
__global__ void ker_attn_softmax_lt32(T *inp, const T *attn_mask, int from_len,
                                      int to_len, bool mask_future) {
  int batch_id = blockIdx.y;
  int head_id = blockIdx.z;
  const int nhead = gridDim.z;
  const int token_per_reduce = 1;
  typedef hipcub::BlockLoad<T, block_dim, ele_per_thread,
                         hipcub::BLOCK_LOAD_VECTORIZE>
      BlockLoad;
  __shared__ typename BlockLoad::TempStorage ts_load;
  typedef hipcub::BlockStore<T, block_dim, ele_per_thread,
                          hipcub::BLOCK_STORE_VECTORIZE>
      BlockStore;
  __shared__ typename BlockStore::TempStorage ts_store;

  T mval[ele_per_thread];
  if (attn_mask) {
    attn_mask += batch_id * to_len;
    BlockLoad(ts_load).Load(attn_mask, mval, to_len, REDUCE_FLOAT_INF_NEG);
  }

  inp += flat_3dim(batch_id, head_id, 0, nhead, from_len * to_len);
  for (int token_id = blockIdx.x * token_per_reduce; token_id < from_len;
       token_id += gridDim.x * token_per_reduce) {
    T inp_val[token_per_reduce][ele_per_thread];
    for (int i = 0; i < token_per_reduce && (token_id + i) < from_len; i++) {
      BlockLoad(ts_load).Load(inp + (token_id + i) * to_len, inp_val[i], to_len,
                              REDUCE_FLOAT_INF_NEG);
    }

    /* step 1. compute max */
    // thread local max
    // Hint: use fmaxf() to compute max
    // BEGIN ASSIGN3_1
    float val[token_per_reduce][ele_per_thread];
    float l_max[token_per_reduce];
    for (int i = 0; i < token_per_reduce; i++) {
      l_max[i] = REDUCE_FLOAT_INF_NEG;
      for (int j = 0; j < ele_per_thread; j++) {
        float temp_val;
        if (mask_future && ele_per_thread * threadIdx.x + j > token_id + i) {
          temp_val = REDUCE_FLOAT_INF_NEG;
        } else {
          temp_val = (float)inp_val[i][j];
          if (attn_mask) {
            temp_val += (float)mval[j];
          }
        }
        val[i][j] = temp_val;
        l_max[i] = fmaxf(l_max[i], temp_val);
      }
    }
    // END ASSIGN3_1
    // warp reduce max
    warpReduce<ReduceType::kMax, token_per_reduce>(l_max);

    /* step 2. compute sum */
    // thread local sum
    // BEGIN ASSIGN3_1
    // Hint: use __expf() to compute exp
    float l_sum[token_per_reduce];
    for (int i = 0; i < token_per_reduce; i++) {
      l_sum[i] = 0.f;
      for (int j = 0; j < ele_per_thread; j++) {
        val[i][j] = __expf(val[i][j] - l_max[i]);
        l_sum[i] += val[i][j];
      }
    }
    // END ASSIGN3_1
    // warp reduce sum
    warpReduce<ReduceType::kSum, token_per_reduce>(l_sum);

    /* step 3. compute final result */
    // BEGIN ASSIGN3_1
    // Hint: use __fdividef() to compute division
    // Hint: use BlockStore to store the result
    for (int i = 0; i < token_per_reduce && (token_id + i) < from_len; i++) {
      l_sum[i] = __fdividef(1.0f, l_sum[i] + EPSILON);
      for (int j = 0; j < ele_per_thread; j++) {
        inp_val[i][j] = (T)(val[i][j] * l_sum[i]);
      }
      BlockStore(ts_store).Store(inp + (token_id + i) * to_len, inp_val[i],
                                 to_len);
    }
    // END ASSIGN3_1
  }  // blockIdx.x
}

template <typename T, int block_dim, int ele_per_thread>
__global__ void ker_attn_softmax(T *inp, const T *attn_mask, int from_len,
                                 int to_len, bool mask_future) {
  
  int batch_id = blockIdx.y;
  int head_id = blockIdx.z;
  const int nhead = gridDim.z;
  const int token_per_reduce = 1;
  typedef hipcub::BlockLoad<T, block_dim, ele_per_thread,
                         hipcub::BLOCK_LOAD_VECTORIZE>
      BlockLoad;
  __shared__ typename BlockLoad::TempStorage ts_load;
  typedef hipcub::BlockStore<T, block_dim, ele_per_thread,
                          hipcub::BLOCK_STORE_VECTORIZE>
      BlockStore;
  __shared__ typename BlockStore::TempStorage ts_store;

  T mval[ele_per_thread];
  if (attn_mask) {
    attn_mask += batch_id * to_len;
    BlockLoad(ts_load).Load(attn_mask, mval, to_len, REDUCE_FLOAT_INF_NEG);
  }

  inp += flat_3dim(batch_id, head_id, 0, nhead, from_len * to_len);
  for (int token_id = blockIdx.x * token_per_reduce; token_id < from_len;
       token_id += gridDim.x * token_per_reduce) {
    T inp_val[token_per_reduce][ele_per_thread];
    for (int i = 0; i < token_per_reduce && (token_id + i) < from_len; i++) {
      BlockLoad(ts_load).Load(inp + (token_id + i) * to_len, inp_val[i], to_len,
                              REDUCE_FLOAT_INF_NEG);
    }

    /* step 1. compute max */
    // thread local max
    // BEGIN ASSIGN3_1
    float val[token_per_reduce][ele_per_thread];
    float l_max[token_per_reduce];
    for (int i = 0; i < token_per_reduce; i++) {
        l_max[i] = REDUCE_FLOAT_INF_NEG;
        for (int j = 0; j < ele_per_thread; j++) {
            float temp_val;
            if (mask_future && ele_per_thread * threadIdx.x + j > token_id + i) {
                temp_val = REDUCE_FLOAT_INF_NEG;
            } else {
                temp_val = (float)inp_val[i][j];
                if (attn_mask) {
                    temp_val += (float)mval[j];
                }
            }
            val[i][j] = temp_val;
            l_max[i] = fmaxf(l_max[i], temp_val);
        }
    }
    // END ASSIGN3_1
    // block reduce max
    blockReduce<ReduceType::kMax, token_per_reduce>(l_max);
    // write shared
    __shared__ float s_max[token_per_reduce];
    if (threadIdx.x == 0) {
      for (int i = 0; i < token_per_reduce; i++) {
        s_max[i] = l_max[i];
      }
    }
    __syncthreads();

    /* step 2. compute sum */
    // thread local sum
    // BEGIN ASSIGN3_1
    float l_sum[token_per_reduce];
    for (int i = 0; i < token_per_reduce; i++) {
        l_sum[i] = 0.f;
        for (int j = 0; j < ele_per_thread; j++) {
            val[i][j] = __expf(val[i][j] - s_max[i]);
            l_sum[i] += val[i][j];
        }
    }
    // END ASSIGN3_1
    // block reduce sum
    blockReduce<ReduceType::kSum, token_per_reduce>(l_sum);
    // write shared
    __shared__ float s_sum[token_per_reduce];
    if (threadIdx.x == 0) {
      for (int i = 0; i < token_per_reduce; i++) {
        s_sum[i] = __fdividef(1.0f, l_sum[i] + EPSILON);
      }
    }
    __syncthreads();

    /* step 3. compute final result */
    // BEGIN ASSIGN3_1
    for (int i = 0; i < token_per_reduce && (token_id + i) < from_len; i++) {
        for (int j = 0; j < ele_per_thread; j++) {
            inp_val[i][j] = (T)(val[i][j] * s_sum[i]);
        }
        BlockStore(ts_store).Store(inp + (token_id + i) * to_len, inp_val[i], to_len);
    }   
    // END ASSIGN3_1
  }  // blockIdx.x
}

/*
  attn_mask!=nullptr for enc-self-attn and enc-dec-attn
  attn_mask=nullptr and mask_future=ture for dec-self-attn training
  attn_mask=nullptr and mask_future=false for dec-self-attn infer
*/
// template <>
extern "C" {
void launch_attn_softmax(float *inp, const float *attn_mask,
                                int batch_size, int nhead, int from_len,
                                int to_len, bool mask_future,
                                hipStream_t stream) {

  int float_size = sizeof(float);
  int inp_size = batch_size * nhead * from_len * to_len * float_size;
  int attn_mask_size = batch_size * to_len * float_size;

  float *d_inp, *d_attn_mask;
  hipMalloc((void **)&d_inp, inp_size);
  hipMalloc((void **)&d_attn_mask, attn_mask_size);

  hipMemcpy(d_inp, inp, inp_size, hipMemcpyHostToDevice);
  hipMemcpy(d_attn_mask, attn_mask, attn_mask_size, hipMemcpyHostToDevice);

  dim3 grid_dim(1, batch_size, nhead);
  if (to_len <= 32) {
    ker_attn_softmax_lt32<float, 32, 1><<<grid_dim, 32, 0, stream>>>(
        d_inp, d_attn_mask, from_len, to_len, mask_future);
  } else if (to_len <= 64) {
    ker_attn_softmax_lt32<float, 32, 2><<<grid_dim, 32, 0, stream>>>(
        d_inp, d_attn_mask, from_len, to_len, mask_future);
  } else if (to_len <= 128) {
    grid_dim.x = 16;
    ker_attn_softmax<float, 64, 2><<<grid_dim, 64, 0, stream>>>(
        d_inp, d_attn_mask, from_len, to_len, mask_future);
  } else if (to_len <= 256) {
    grid_dim.x = 32;
    ker_attn_softmax<float, 128, 2><<<grid_dim, 128, 0, stream>>>(
        d_inp, d_attn_mask, from_len, to_len, mask_future);
  } else if (to_len <= 512) {
    grid_dim.x = 64;
    ker_attn_softmax<float, 256, 2><<<grid_dim, 256, 0, stream>>>(
        d_inp, d_attn_mask, from_len, to_len, mask_future);
  } else if (to_len <= 1024) {
    grid_dim.x = 128;
    ker_attn_softmax<float, 512, 2><<<grid_dim, 512, 0, stream>>>(
        d_inp, d_attn_mask, from_len, to_len, mask_future);
  } else {
    throw std::runtime_error(
        "Sequence length greater than 512 is currently not supported");
  }

  // Copy back to the host
  hipMemcpy(inp, d_inp, inp_size, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  // Check CUDA execution
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "launch_attn_softmax Error: %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Free memory on device
  hipFree(d_inp);
  hipFree(d_attn_mask);

}}


/**
@brief: ker_attn_softmax_bw
Softmax backward in self attention.

@thread
gridDim.x = batch_size * nhead * seq_len / warps_per_block
blockDim.x = WARP_SIZE
blockDim.y = warps_per_block = 4

@param
grad: [batch_size, nhead, seq_len, seq_len], output grad.
output: [batch_size, nhead, seq_len, seq_len], output of softmax forward.
*/
template <typename T, int ITERATIONS>
__global__ void ker_attn_softmax_bw(T *grad, const T *inp, int softmax_length) {
  int batch_idx = blockIdx.x * blockDim.y + threadIdx.y; // row
  int offset = batch_idx * softmax_length + threadIdx.x; // 32 threads work on same row

  grad += offset;
  inp += offset;

  // ITERATIONS = ceil(softmax_len / 32)
  T grad_reg[ITERATIONS];
  T inp_reg[ITERATIONS];
  float sum = 0.0;

  // dot product sum_j (partial L)/(partial sigma_j) * sigma_j
  // strided access => coalesce memory access
  #pragma unroll // unroll/expand loop => more efficient (e.g. eliminate loop overhead)
  for (int i = 0; i < ITERATIONS; ++i) {
    int curr_idx = threadIdx.x + i * WARP_SIZE;
    if (curr_idx < softmax_length) {
      grad_reg[i] = grad[i * WARP_SIZE];
      inp_reg[i] = inp[i * WARP_SIZE];
      sum += (float)grad_reg[i] * (float)inp_reg[i];
    }
  }

  cg::thread_block b = cg::this_thread_block(); // handle to current thread block
  cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b); // divide thread block into group of threads

  // warp level sum (butterfly reduction)
  for (int i = 1; i < WARP_SIZE; i <<= 1) sum += g.shfl_xor(sum, i);

  // gradient formula
  #pragma unroll
  for (int i = 0; i < ITERATIONS; ++i) {
    int curr_idx = threadIdx.x + i * WARP_SIZE;
    if (curr_idx < softmax_length)
      grad[i * WARP_SIZE] = (T)((float)inp_reg[i] * ((float)grad_reg[i] - sum));
  }
}

// template <typename T>
extern "C" {
void launch_attn_softmax_bw(float *out_grad,
                                const float *soft_inp, int rows,
                                int softmax_len,
                                hipStream_t stream) {
  
  const int warps_per_block = 4;
  dim3 grid_dim((rows + warps_per_block - 1) / warps_per_block);
  dim3 block_dim(WARP_SIZE, warps_per_block);

  int float_size = sizeof(float);
  int out_grad_size = rows * softmax_len * float_size;
  int soft_inp_size = rows * softmax_len * float_size;

  float *d_out_grad, *d_soft_inp;
  hipMalloc((void **)&d_out_grad, out_grad_size);
  hipMalloc((void **)&d_soft_inp, soft_inp_size);

  hipMemcpy(d_out_grad, out_grad, out_grad_size, hipMemcpyHostToDevice);
  hipMemcpy(d_soft_inp, soft_inp, soft_inp_size, hipMemcpyHostToDevice);
  
  if (softmax_len <= 32) {
      ker_attn_softmax_bw<float, 1><<<grid_dim, block_dim, 0, stream>>>(
          d_out_grad, d_soft_inp, softmax_len);
  } else if (softmax_len <= 64) {
      ker_attn_softmax_bw<float, 2><<<grid_dim, block_dim, 0, stream>>>(
          d_out_grad, d_soft_inp, softmax_len);
  } else if (softmax_len <= 128) {
      ker_attn_softmax_bw<float, 4><<<grid_dim, block_dim, 0, stream>>>(
          d_out_grad, d_soft_inp, softmax_len);
  } else if (softmax_len <= 256) {
      ker_attn_softmax_bw<float, 8><<<grid_dim, block_dim, 0, stream>>>(
          d_out_grad, d_soft_inp, softmax_len);
  } else if (softmax_len <= 384) {
      ker_attn_softmax_bw<float, 12><<<grid_dim, block_dim, 0, stream>>>(
          d_out_grad, d_soft_inp, softmax_len);
  } else if (softmax_len <= 512) {
      ker_attn_softmax_bw<float, 16><<<grid_dim, block_dim, 0, stream>>>(
          d_out_grad, d_soft_inp, softmax_len);
  } else if (softmax_len <= 768) {
      ker_attn_softmax_bw<float, 24><<<grid_dim, block_dim, 0, stream>>>(
          d_out_grad, d_soft_inp, softmax_len);
  } else if (softmax_len <= 1024) {
      ker_attn_softmax_bw<float, 32><<<grid_dim, block_dim, 0, stream>>>(
          d_out_grad, d_soft_inp, softmax_len);
  } else if (softmax_len <= 2048) {
      ker_attn_softmax_bw<float, 64><<<grid_dim, block_dim, 0, stream>>>(
          d_out_grad, d_soft_inp, softmax_len);
  } else {
      throw std::runtime_error("Unsupported max sequence length");
  }

  hipMemcpy(out_grad, d_out_grad, out_grad_size, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "launch_attn_softmax_bw Error: %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  hipFree(d_out_grad);
  hipFree(d_soft_inp);
}}

}  
} 
