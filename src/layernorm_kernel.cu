#include "hip/hip_runtime.h"
#include "includes/block_reduce.h"
#include "includes/kernels.h"
#include "includes/cuda_util.h"

#include <hip/hip_cooperative_groups.h>
#include <cstddef>

namespace cg = cooperative_groups;
namespace lightseq {
namespace cuda {

const float LN_EPSILON = 1e-8f;
#define TILE_DIM 32


/**
@brief: ker_layer_norm
Standard layer normalization.
It will not only output the layer norm result,
  but also outputs variance.
  may also output means, depends on whether
  the means argument is nullptr

@thread
gridDim.x = batch_size * seq_len
blockDim.x = hidden_size

@param
ln_res: [batch_size * seq_len, hidden_dim], ln result.
vars: [batch_size * seq_len], variance per token
means: [batch_size * seq_len], means per token, can be nullput
inp: [batch_size * seq_len, hidden_dim], ln input.
scale: [hidden_dim], ln scale
bias: [hidden_dim], ln bias
*/
template <typename T>
__global__ void ker_layer_norm(T *ln_res, T *vars, T *means, const T *inp,
                               const T *scale, const T *bias, int hidden_size) {
  
  /// BEGIN ASSIGN3_2
  /// TODO
  // Hints:
  // 1. Compute x and x^2 with reinterpret_cast by casting to float4 for speedup
  // 2. Compute reduce sum with blockReduce and add epsilon with LN_EPSILON
  // 3. Compute layernorm result with reinterpret_cast by casting to float4 for speedup

  // NOTE: hidden_size is hidden_dim / 4 
  
  // Step 1
  float l_sum = 0;
  float l_sq_sum = 0;
  const float4 *inp_f4 = reinterpret_cast<const float4 *>(inp) + blockIdx.x * hidden_size;  
  for (uint idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    float4 val = inp_f4[idx];
    l_sum += val.x + val.y + val.z + val.w;
    l_sq_sum += val.x * val.x + val.y * val.y + val.z * val.z + val.w * val.w;
  }

  // Step 2
  float reduce_vals[2] = {l_sum, l_sq_sum};
  blockReduce<ReduceType::kSum, 2>(reduce_vals);

  __shared__ float s_mean, s_var;
  int h_dim = hidden_size * 4;
  if (threadIdx.x == 0) {
    s_mean = reduce_vals[0] / h_dim;
    if (means != nullptr) {
      means[blockIdx.x] = s_mean;
    }

    s_var = (reduce_vals[1] / h_dim) - s_mean * s_mean + LN_EPSILON;
    vars[blockIdx.x] = s_var;

    s_var = rsqrtf(s_var); // s_var is now 1/stdev
  }
  __syncthreads(); // important! 

  // Step 3
  float4 *out_f4 = reinterpret_cast<float4 *>(ln_res) + blockIdx.x * hidden_size;
  for (uint idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    float4 scale_f4 = *(reinterpret_cast<const float4 *>(scale) + idx);
    float4 bias_f4 = *(reinterpret_cast<const float4 *>(bias) + idx);

    float4 inp_val = inp_f4[idx];

    inp_val.x = (inp_val.x - s_mean) * s_var * scale_f4.x + bias_f4.x;
    inp_val.y = (inp_val.y - s_mean) * s_var * scale_f4.y + bias_f4.y;
    inp_val.z = (inp_val.z - s_mean) * s_var * scale_f4.z + bias_f4.z;
    inp_val.w = (inp_val.w - s_mean) * s_var * scale_f4.w + bias_f4.w;

    out_f4[idx] = inp_val;
  }
  /// END ASSIGN3_2
}

extern "C" {
void launch_layernorm(float *ln_res, float *vars, float *means,
                              const float *inp, const float *scale,
                              const float *bias, int batch_size, int hidden_dim,
                              hipStream_t stream) {
  if (hidden_dim % 4 != 0) {
    throw std::runtime_error("violate hidden_dim % 4 = 0");
  }
  int float_size = sizeof(float);
  int input_size = batch_size * hidden_dim * float_size;
  int scale_size = hidden_dim * float_size;
  int bias_size = hidden_dim * float_size;
  int output_size = batch_size * hidden_dim * float_size;
  int mean_size = batch_size * float_size;
  int var_size = batch_size * float_size;


  float *d_ln_res, *d_vars, *d_means, *d_inp, *d_scale, *d_bias;
  hipMalloc((void **)&d_ln_res, output_size);
  hipMalloc((void **)&d_vars, var_size);
  hipMalloc((void **)&d_means, mean_size);
  hipMalloc((void **)&d_inp, input_size);
  hipMalloc((void **)&d_scale, scale_size);
  hipMalloc((void **)&d_bias, bias_size);

  hipMemcpy(d_inp, inp, input_size, hipMemcpyHostToDevice);
  hipMemcpy(d_scale, scale, scale_size, hipMemcpyHostToDevice);
  hipMemcpy(d_bias, bias, bias_size, hipMemcpyHostToDevice);

  // For using float4
  hidden_dim >>= 2;
  int nthread = min(((hidden_dim + 31) / 32) * 32, MAX_THREADS);
  dim3 grid_dim(batch_size);
  dim3 block_dim(nthread);

  ker_layer_norm<float><<<grid_dim, block_dim, 0, stream>>>(
      d_ln_res, d_vars, d_means, d_inp, d_scale, d_bias, hidden_dim);

  // Copy back to the host
  hipMemcpy(ln_res, d_ln_res, output_size, hipMemcpyDeviceToHost);
  hipMemcpy(vars, d_vars, var_size, hipMemcpyDeviceToHost);
  hipMemcpy(means, d_means, mean_size, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  // Check CUDA execution
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "launch_layernorm Error: %s\n", hipGetErrorString(err));
    // Handle the error (e.g., by exiting the program)
    exit(EXIT_FAILURE);
  }

  // Free memory on device
  hipFree(d_ln_res);
  hipFree(d_vars);
  hipFree(d_means);
  hipFree(d_inp);
  hipFree(d_scale);
  hipFree(d_bias);

}
}

/**
@brief: ker_ln_bw_dgamma_dbetta
Layer norm backword kernel, compute the gradient of gamma and betta.
dbetta = sum(dout, dim=0)
dgamma = sum(xhat * dout, dim=0)
xhat = (input - mean) * rsqrt(var) or
  (output - betta) / gamma

@thread
gridDim.x = hidden_size / 32
blockDim.x = 32
blockDim.y = 32

@param
gamma_grad: [hidden_size], gradient of gamma
betta_grad: [hidden_size], gradient of betta
out_grad: [batch_size * seq_len, hidden_size], gradient of betta ln output
inp_or_out: [batch_size * seq_len, hidden_size], ln output if means is nullptr
  ln input if means is not nullptr
gamma: [hidden_size], gamma of ln,
  used to compute xhat, maybe nullptr
betta: [hidden_size], betta of ln,
  used to compute xhat, maybe nullptr
vars: [batch_size * seq_len], variance of ln forward,
  used to compute xhat, maybe nullptr
means: [batch_size * seq_len], mean of ln forward,
  used to compute xhat, maybe nullptr
(gamma && betta) ^ (vars && means) should be true
*/
template <typename T>
__global__ void ker_ln_bw_dgamma_dbetta(T *gamma_grad, T *betta_grad,
                                        const T *out_grad,
                                        const T *inp, const T *gamma,
                                        const T *betta, const T *vars,
                                        const T *means, int rows, int width) {

  /// BEGIN ASSIGN3_2
  /// TODO
  // Hints:
  // 1. Compute the partial gradients by looping across inp rows
  // 2. Store the partial gradients in the shared memory arrays
  // 3. Compute the reduce sum of the shared memory arrays with g.shfl_down
  //      -> More hints about `g.shfl_down`:
  //      -> https://developer.nvidia.com/blog/cooperative-groups/#:~:text=Using%20thread_block_tile%3A%3Ashfl_down()%20to%20simplify%20our%20warp%2Dlevel%20reduction%20does%20benefit%20our%20code%3A%20it%20simplifies%20it%20and%20eliminates%20the%20need%20for%20shared%20memory
  //      -> The highlighted line gives you a conceptual understanding of what the g.shfl_down is doing. Usually, the threads inside a block need to load everything to shared memory and work together to reduce the result (like what you have implemented in the hw1 for reduce function). 
  //      -> Now g.shfl_down helps you do so without consuming any shared memory. g.shfl_down makes it more efficient.
  // 4. Assign the final result to the correct position in the global output

  __shared__ float betta_buffer[TILE_DIM][TILE_DIM];
  __shared__ float gamma_buffer[TILE_DIM][TILE_DIM];

  cg::thread_block b = cg::this_thread_block();
  cg::thread_block_tile<TILE_DIM> g = cg::tiled_partition<TILE_DIM>(b);

  // Step 1

  // Step 2
  
  // Step 3
  
  // Step 4

  assert(false && "Not Implemented");
  /// END ASSIGN3_2
}

/**
@brief: ker_ln_bw_dinp
Layer norm backword kernel, compute the gradient of input.
dinp = (dxhat - (sum(dxhat) + xhat * sum(dxhat * xhat)) / hidden_dim)
  * rsqrt(var)
xhat = (input - mean) * rsqrt(var) if mean is not nullptr
       (output - betta) / gamma if mean is nullptr
dxhat = dout * gamma


@thread
gridDim.x = batch_size * seq_len
blockDim.x = hidden_size

@param
inp_grad: [batch_size * seq_len, hidden_size], gradient of betta ln output
out_grad: [batch_size * seq_len, hidden_size], gradient of betta ln output
residual_grad: [batch_size * seq_len, hidden_size], gradient of residual input,
  usually appear in pre-layer-norm for transformer layer, maybe nullptr
inp_or_out: [batch_size * seq_len, hidden_size], ln output if means is nullptr
  ln input if means is not nullptr
gamma: [hidden_size], gamma of ln,
  used to compute xhat and dxhat
betta: [hidden_size], betta of ln,
  used to compute xhat, maybe nullptr
vars: [batch_size * seq_len], variance of ln forward,
  used to compute xhat and dinp
means: [batch_size * seq_len], mean of ln forward,
  used to compute xhat, maybe nullptr
*/
template <typename T>
__global__ void ker_ln_bw_dinp(T *inp_grad, const T *out_grad, const T *inp,
                               const T *gamma, const T *betta, const T *vars,
                               const T *means, int hidden_dim) {
  
  /// BEGIN ASSIGN3_2
  /// TODO
  // Hints:
  // 1. Compute dxhat=dy*w with reinterpret_cast by casting to float4 for speedup
  // 2. Compute xhat with reinterpret_cast by casting to float4 for speedup
  // 3. Compute reduce sum for dxhat and dxhat*xhat with blockReduce
  // 4. Compute final gradient
  
  // Step 1
 
  // Step 2
   
  // Step 3
 
  // Step 4
  
  assert(false && "Not Implemented");
  /// END ASSIGN3_2
}
extern "C" {
void launch_layernorm_bw(float *gamma_grad, float *betta_grad, float *inp_grad,
                         const float *out_grad, const float *inp, const float *gamma,
                         const float *betta, const float *vars,
                         const float *means, int batch_size, int hidden_dim,
                         hipStream_t stream_1, hipStream_t stream_2) {
  
  // Allocate device memory
  float *d_gamma_grad, *d_betta_grad, *d_inp_grad, *d_out_grad, *d_inp, *d_gamma, *d_betta, *d_vars, *d_means;
  int grad_output_size = batch_size * hidden_dim * sizeof(float);
  int gamma_betta_size = hidden_dim * sizeof(float);
  int vars_means_size = batch_size * sizeof(float);

  hipMalloc((void **)&d_gamma_grad, gamma_betta_size);
  hipMalloc((void **)&d_betta_grad, gamma_betta_size);
  hipMalloc((void **)&d_inp_grad, grad_output_size);
  hipMalloc((void **)&d_out_grad, grad_output_size);
  hipMalloc((void **)&d_inp, grad_output_size);
  hipMalloc((void **)&d_gamma, gamma_betta_size);
  hipMalloc((void **)&d_betta, gamma_betta_size);
  hipMalloc((void **)&d_vars, vars_means_size);
  hipMalloc((void **)&d_means, vars_means_size);

  // Copy memory to device
  hipMemcpy((void *)d_out_grad, out_grad, grad_output_size, hipMemcpyHostToDevice);
  hipMemcpy((void *)d_inp, inp, grad_output_size, hipMemcpyHostToDevice);
  hipMemcpy((void *)d_gamma, gamma, gamma_betta_size, hipMemcpyHostToDevice);
  hipMemcpy((void *)d_betta, betta, gamma_betta_size, hipMemcpyHostToDevice);
  hipMemcpy((void *)d_vars, vars, vars_means_size, hipMemcpyHostToDevice);
  hipMemcpy((void *)d_means, means, vars_means_size, hipMemcpyHostToDevice);

  // Launch kernels
  // Compute grad of gamma and betta
  // This calculates the number of blocks needed to cover the data along the specified dimension, rounds it up.
  // The result is then multiplied by TILE_DIM to ensure that the grid size is a multiple of TILE_DIM.
  dim3 grid_dim(((hidden_dim + TILE_DIM - 1) / TILE_DIM) * TILE_DIM);
  dim3 block_dim(TILE_DIM, TILE_DIM);
  ker_ln_bw_dgamma_dbetta<float><<<grid_dim, block_dim, 0, stream_1>>>(
      d_gamma_grad, d_betta_grad, d_out_grad, d_inp, d_gamma, d_betta, d_vars,
      d_means, batch_size, hidden_dim);

  // Compute grad of input
  if (hidden_dim % 4 != 0 || hidden_dim > 4096) {
    throw std::runtime_error("hidden_dim % 4 != 0 || hidden_dim > 4096");
  }
  hidden_dim >>= 2;
  int nthread = min(((hidden_dim + 31) / 32) * 32, MAX_THREADS);
  ker_ln_bw_dinp<<<batch_size, nthread, 0, stream_2>>>(
      d_inp_grad, d_out_grad, d_inp, d_gamma, d_betta, d_vars, d_means, hidden_dim);

  // Synchronize and check for errors
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "launch_layernorm_bw Error: %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Copy back to host
  hipMemcpy(gamma_grad, d_gamma_grad, gamma_betta_size, hipMemcpyDeviceToHost);
  hipMemcpy(betta_grad, d_betta_grad, gamma_betta_size, hipMemcpyDeviceToHost);
  hipMemcpy(inp_grad, d_inp_grad, grad_output_size, hipMemcpyDeviceToHost);

  // Free device memory
  hipFree(d_gamma_grad);
  hipFree(d_betta_grad);
  hipFree(d_inp_grad);
  hipFree((void *)d_out_grad);
  hipFree((void *)d_inp);
  hipFree((void *)d_gamma);
  hipFree((void *)d_betta);
  hipFree((void *)d_vars);
  hipFree((void *)d_means);
}}
}}
