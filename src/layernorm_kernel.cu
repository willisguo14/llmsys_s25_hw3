#include "hip/hip_runtime.h"
#include "includes/block_reduce.h"
#include "includes/kernels.h"
#include "includes/cuda_util.h"

#include <hip/hip_cooperative_groups.h>
#include <cstddef>

namespace cg = cooperative_groups;
namespace lightseq {
namespace cuda {

const float LN_EPSILON = 1e-8f;
#define TILE_DIM 32



/**
@brief: ker_layer_norm
Standard layer normalization.
It will not only output the layer norm result,
  but also outputs variance.
  may also output means, depends on whether
  the means argument is nullptr

@thread
gridDim.x = batch_size * seq_len
blockDim.x = hidden_size

@param
ln_res: [batch_size * seq_len, hidden_dim], ln result.
vars: [batch_size * seq_len], variance per token
means: [batch_size * seq_len], means per token, can be nullput
inp: [batch_size * seq_len, hidden_dim], ln input.
scale: [hidden_dim], ln scale
bias: [hidden_dim], ln bias
*/
template <typename T>
__global__ void ker_layer_norm(T *ln_res, T *vars, T *means, const T *inp,
                               const T *scale, const T *bias, int hidden_size) {
  
  /// BEGIN ASSIGN3_2
  /// TODO
  // Hints:
  // 1. Compute x and x^2 with reinterpret_cast by casting to float4 for speedup
  // 2. Compute reduce sum with blockReduce and add epsilon with LN_EPSILON
  // 3. Compute layernorm result with reinterpret_cast by casting to float4 for speedup

  // NOTE: hidden_size is hidden_dim / 4 
  
  // Step 1
  float l_sum = 0;
  float l_sq_sum = 0;
  const float4 *inp_f4 = reinterpret_cast<const float4 *>(inp) + blockIdx.x * hidden_size;  
  for (uint idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    float4 val = inp_f4[idx];
    l_sum += val.x + val.y + val.z + val.w;
    l_sq_sum += val.x * val.x + val.y * val.y + val.z * val.z + val.w * val.w;
  }

  // Step 2
  float reduce_vals[2] = {l_sum, l_sq_sum};
  blockReduce<ReduceType::kSum, 2>(reduce_vals);

  __shared__ float s_mean, s_var;
  int h_dim = hidden_size * 4;
  if (threadIdx.x == 0) {
    s_mean = reduce_vals[0] / h_dim;
    if (means != nullptr) {
      means[blockIdx.x] = s_mean;
    }

    s_var = (reduce_vals[1] / h_dim) - s_mean * s_mean + LN_EPSILON;
    vars[blockIdx.x] = s_var;

    s_var = rsqrtf(s_var); // s_var is now 1/stdev
  }
  __syncthreads(); // important! 

  // Step 3
  float4 *out_f4 = reinterpret_cast<float4 *>(ln_res) + blockIdx.x * hidden_size;
  for (uint idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    float4 scale_f4 = *(reinterpret_cast<const float4 *>(scale) + idx);
    float4 bias_f4 = *(reinterpret_cast<const float4 *>(bias) + idx);

    float4 inp_val = inp_f4[idx];

    inp_val.x = (inp_val.x - s_mean) * s_var * scale_f4.x + bias_f4.x;
    inp_val.y = (inp_val.y - s_mean) * s_var * scale_f4.y + bias_f4.y;
    inp_val.z = (inp_val.z - s_mean) * s_var * scale_f4.z + bias_f4.z;
    inp_val.w = (inp_val.w - s_mean) * s_var * scale_f4.w + bias_f4.w;

    out_f4[idx] = inp_val;
  }
  /// END ASSIGN3_2
}

extern "C" {
void launch_layernorm(float *ln_res, float *vars, float *means,
                              const float *inp, const float *scale,
                              const float *bias, int batch_size, int hidden_dim,
                              hipStream_t stream) {
  if (hidden_dim % 4 != 0) {
    throw std::runtime_error("violate hidden_dim % 4 = 0");
  }
  int float_size = sizeof(float);
  int input_size = batch_size * hidden_dim * float_size;
  int scale_size = hidden_dim * float_size;
  int bias_size = hidden_dim * float_size;
  int output_size = batch_size * hidden_dim * float_size;
  int mean_size = batch_size * float_size;
  int var_size = batch_size * float_size;


  float *d_ln_res, *d_vars, *d_means, *d_inp, *d_scale, *d_bias;
  hipMalloc((void **)&d_ln_res, output_size);
  hipMalloc((void **)&d_vars, var_size);
  hipMalloc((void **)&d_means, mean_size);
  hipMalloc((void **)&d_inp, input_size);
  hipMalloc((void **)&d_scale, scale_size);
  hipMalloc((void **)&d_bias, bias_size);

  hipMemcpy(d_inp, inp, input_size, hipMemcpyHostToDevice);
  hipMemcpy(d_scale, scale, scale_size, hipMemcpyHostToDevice);
  hipMemcpy(d_bias, bias, bias_size, hipMemcpyHostToDevice);

  // For using float4
  hidden_dim >>= 2;
  int nthread = min(((hidden_dim + 31) / 32) * 32, MAX_THREADS);
  dim3 grid_dim(batch_size);
  dim3 block_dim(nthread);

  ker_layer_norm<float><<<grid_dim, block_dim, 0, stream>>>(
      d_ln_res, d_vars, d_means, d_inp, d_scale, d_bias, hidden_dim);

  // Copy back to the host
  hipMemcpy(ln_res, d_ln_res, output_size, hipMemcpyDeviceToHost);
  hipMemcpy(vars, d_vars, var_size, hipMemcpyDeviceToHost);
  hipMemcpy(means, d_means, mean_size, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  // Check CUDA execution
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "launch_layernorm Error: %s\n", hipGetErrorString(err));
    // Handle the error (e.g., by exiting the program)
    exit(EXIT_FAILURE);
  }

  // Free memory on device
  hipFree(d_ln_res);
  hipFree(d_vars);
  hipFree(d_means);
  hipFree(d_inp);
  hipFree(d_scale);
  hipFree(d_bias);

}
}

/**
@brief: ker_ln_bw_dgamma_dbetta
Layer norm backword kernel, compute the gradient of gamma and betta.
dbetta = sum(dout, dim=0)
dgamma = sum(xhat * dout, dim=0)
xhat = (input - mean) * rsqrt(var) or
  (output - betta) / gamma

@thread
gridDim.x = hidden_size / 32
blockDim.x = 32
blockDim.y = 32

@param
gamma_grad: [hidden_size], gradient of gamma
betta_grad: [hidden_size], gradient of betta
out_grad: [batch_size * seq_len, hidden_size], gradient of betta ln output
inp_or_out: [batch_size * seq_len, hidden_size], ln output if means is nullptr
  ln input if means is not nullptr
gamma: [hidden_size], gamma of ln,
  used to compute xhat, maybe nullptr
betta: [hidden_size], betta of ln,
  used to compute xhat, maybe nullptr
vars: [batch_size * seq_len], variance of ln forward,
  used to compute xhat, maybe nullptr
means: [batch_size * seq_len], mean of ln forward,
  used to compute xhat, maybe nullptr
(gamma && betta) ^ (vars && means) should be true
*/
template <typename T>
__global__ void ker_ln_bw_dgamma_dbetta(T *gamma_grad, T *betta_grad,
                                        const T *out_grad,
                                        const T *inp, const T *gamma,
                                        const T *betta, const T *vars,
                                        const T *means, int rows, int width) {

  /// BEGIN ASSIGN3_2
  /// TODO
  // Hints:
  // 1. Compute the partial gradients by looping across inp rows
  // 2. Store the partial gradients in the shared memory arrays
  // 3. Compute the reduce sum of the shared memory arrays with g.shfl_down
  //      -> More hints about `g.shfl_down`:
  //      -> https://developer.nvidia.com/blog/cooperative-groups/#:~:text=Using%20thread_block_tile%3A%3Ashfl_down()%20to%20simplify%20our%20warp%2Dlevel%20reduction%20does%20benefit%20our%20code%3A%20it%20simplifies%20it%20and%20eliminates%20the%20need%20for%20shared%20memory
  //      -> The highlighted line gives you a conceptual understanding of what the g.shfl_down is doing. Usually, the threads inside a block need to load everything to shared memory and work together to reduce the result (like what you have implemented in the hw1 for reduce function). 
  //      -> Now g.shfl_down helps you do so without consuming any shared memory. g.shfl_down makes it more efficient.
  // 4. Assign the final result to the correct position in the global output

  __shared__ float betta_buffer[TILE_DIM][TILE_DIM];
  __shared__ float gamma_buffer[TILE_DIM][TILE_DIM];

  cg::thread_block b = cg::this_thread_block();
  cg::thread_block_tile<TILE_DIM> g = cg::tiled_partition<TILE_DIM>(b);

  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  
  int offset = threadIdx.y * width + idx;
  int offset_stride = width * TILE_DIM;

  // loop across rows
  float dbetta = 0.0f, dgamma = 0.0f;
  float dout, val;

  if (idx < width) {
    for (int r = threadIdx.y; r < rows; r += TILE_DIM) {
      dout = static_cast<float>(out_grad[offset]);
      val = static_cast<float>(inp[offset]); // always inp, not inp_or_out
      
      dgamma += dout * ((val - static_cast<float>(means[r])) * rsqrtf(static_cast<float>(vars[r]) + LN_EPSILON)); // calculate xhat from inp
      dbetta += dout;
      
      offset += offset_stride;
    }
  }

  // sum
  // transpose
  // want to sum buffers along row 
  betta_buffer[threadIdx.x][threadIdx.y] = dbetta;
  gamma_buffer[threadIdx.x][threadIdx.y] = dgamma;
  __syncthreads();

  // threads in same warp (consecutive threadIdx.x) = row
  float betta_sum = betta_buffer[threadIdx.y][threadIdx.x];
  float gamma_sum = gamma_buffer[threadIdx.y][threadIdx.x];

  // shfl_down to share data within partition without using shared memory
  // down-sweep or up-sweep
  for (int i = TILE_DIM/2; i > 0; i >>= 1) {
    betta_sum += g.shfl_down(betta_sum, i);
    gamma_sum += g.shfl_down(gamma_sum, i);
  }

  // after transpose, tx == 0 has the sum for idx ty along h 
  int pos = blockDim.x * blockIdx.x + threadIdx.y;
  if (threadIdx.x == 0 && idx < width) {
    betta_grad[pos] = betta_sum;
    gamma_grad[pos] = gamma_sum;
  }
  /// END ASSIGN3_2
}

/**
@brief: ker_ln_bw_dinp
Layer norm backword kernel, compute the gradient of input.
dinp = (dxhat - (sum(dxhat) + xhat * sum(dxhat * xhat)) / hidden_dim)
  * rsqrt(var)
xhat = (input - mean) * rsqrt(var) if mean is not nullptr
       (output - betta) / gamma if mean is nullptr
dxhat = dout * gamma


@thread
gridDim.x = batch_size * seq_len
blockDim.x = hidden_size

@param
inp_grad: [batch_size * seq_len, hidden_size], gradient of betta ln output
out_grad: [batch_size * seq_len, hidden_size], gradient of betta ln output
residual_grad: [batch_size * seq_len, hidden_size], gradient of residual input,
  usually appear in pre-layer-norm for transformer layer, maybe nullptr
inp_or_out: [batch_size * seq_len, hidden_size], ln output if means is nullptr
  ln input if means is not nullptr
gamma: [hidden_size], gamma of ln,
  used to compute xhat and dxhat
betta: [hidden_size], betta of ln,
  used to compute xhat, maybe nullptr
vars: [batch_size * seq_len], variance of ln forward,
  used to compute xhat and dinp
means: [batch_size * seq_len], mean of ln forward,
  used to compute xhat, maybe nullptr
*/
template <typename T>
__global__ void ker_ln_bw_dinp(T *inp_grad, const T *out_grad, const T *inp,
                               const T *gamma, const T *betta, const T *vars,
                               const T *means, int hidden_dim) {
  
  /// BEGIN ASSIGN3_2
  /// TODO
  // if (threadIdx.x < hidden_dim) return at start? no because blockReduce need ignore threads to have value 0

  // Hints:
  // 1. Compute dxhat=dy*w with reinterpret_cast by casting to float4 for speedup
  // 2. Compute xhat with reinterpret_cast by casting to float4 for speedup
  // 3. Compute reduce sum for dxhat and dxhat*xhat with blockReduce
  // 4. Compute final gradient
  int offset_f4 = blockIdx.x * hidden_dim + threadIdx.x;
  
  float4 dxhat, xhat;
  float rsqrt_var;
  if (threadIdx.x < hidden_dim) {
    // d_xhat = d_out * gamma
    dxhat = reinterpret_cast<const float4 *>(out_grad)[offset_f4]; 
    float4 gamma_f4 = reinterpret_cast<const float4 *>(gamma)[threadIdx.x]; 

    dxhat.x *= gamma_f4.x;
    dxhat.y *= gamma_f4.y;
    dxhat.z *= gamma_f4.z;
    dxhat.w *= gamma_f4.w;

    // xhat = (inp - mean) * rsqrtf(var)
    xhat = reinterpret_cast<const float4 *>(inp)[offset_f4]; // not inp_or_out
    rsqrt_var = rsqrtf(static_cast<float>(vars[blockIdx.x]) + LN_EPSILON);

    float mean = static_cast<float>(means[blockIdx.x]);

    xhat.x = (xhat.x - mean) * rsqrt_var;
    xhat.y = (xhat.y - mean) * rsqrt_var;
    xhat.z = (xhat.z - mean) * rsqrt_var;
    xhat.w = (xhat.w - mean) * rsqrt_var;
  }

  // sums
  float reduce_vals[2] = {0.0f, 0.0f};
  if (threadIdx.x < hidden_dim) {
    reduce_vals[0] = dxhat.x + dxhat.y + dxhat.z + dxhat.w;
    reduce_vals[1] = dxhat.x * xhat.x + dxhat.y * xhat.y + dxhat.z * xhat.z + dxhat.w * xhat.w;
  }
  blockReduce<ReduceType::kSum, 2>(reduce_vals);

  __shared__ float s_sum_dxhat, s_sum_dxhat_xhat;
  if (threadIdx.x == 0) {
    float dim = hidden_dim * 4;
    s_sum_dxhat = reduce_vals[0] / dim;
    s_sum_dxhat_xhat = reduce_vals[1] / dim;
  }
  __syncthreads();

  if (threadIdx.x < hidden_dim) {
    // dxhat is now calculating inp_grad
    dxhat.x = (dxhat.x - s_sum_dxhat - xhat.x * s_sum_dxhat_xhat) * rsqrt_var;
    dxhat.y = (dxhat.y - s_sum_dxhat - xhat.y * s_sum_dxhat_xhat) * rsqrt_var;
    dxhat.z = (dxhat.z - s_sum_dxhat - xhat.z * s_sum_dxhat_xhat) * rsqrt_var;
    dxhat.w = (dxhat.w - s_sum_dxhat - xhat.w * s_sum_dxhat_xhat) * rsqrt_var;

    reinterpret_cast<float4 *>(inp_grad)[offset_f4] = dxhat;
  }
  /// END ASSIGN3_2
}
extern "C" {
void launch_layernorm_bw(float *gamma_grad, float *betta_grad, float *inp_grad,
                         const float *out_grad, const float *inp, const float *gamma,
                         const float *betta, const float *vars,
                         const float *means, int batch_size, int hidden_dim,
                         hipStream_t stream_1, hipStream_t stream_2) {
  
  // Allocate device memory
  float *d_gamma_grad, *d_betta_grad, *d_inp_grad, *d_out_grad, *d_inp, *d_gamma, *d_betta, *d_vars, *d_means;
  int grad_output_size = batch_size * hidden_dim * sizeof(float);
  int gamma_betta_size = hidden_dim * sizeof(float);
  int vars_means_size = batch_size * sizeof(float);

  hipMalloc((void **)&d_gamma_grad, gamma_betta_size);
  hipMalloc((void **)&d_betta_grad, gamma_betta_size);
  hipMalloc((void **)&d_inp_grad, grad_output_size);
  hipMalloc((void **)&d_out_grad, grad_output_size);
  hipMalloc((void **)&d_inp, grad_output_size);
  hipMalloc((void **)&d_gamma, gamma_betta_size);
  hipMalloc((void **)&d_betta, gamma_betta_size);
  hipMalloc((void **)&d_vars, vars_means_size);
  hipMalloc((void **)&d_means, vars_means_size);

  // Copy memory to device
  hipMemcpy((void *)d_out_grad, out_grad, grad_output_size, hipMemcpyHostToDevice);
  hipMemcpy((void *)d_inp, inp, grad_output_size, hipMemcpyHostToDevice);
  hipMemcpy((void *)d_gamma, gamma, gamma_betta_size, hipMemcpyHostToDevice);
  hipMemcpy((void *)d_betta, betta, gamma_betta_size, hipMemcpyHostToDevice);
  hipMemcpy((void *)d_vars, vars, vars_means_size, hipMemcpyHostToDevice);
  hipMemcpy((void *)d_means, means, vars_means_size, hipMemcpyHostToDevice);

  // Launch kernels
  // Compute grad of gamma and betta
  // This calculates the number of blocks needed to cover the data along the specified dimension, rounds it up.
  // The result is then multiplied by TILE_DIM to ensure that the grid size is a multiple of TILE_DIM.
  dim3 grid_dim(((hidden_dim + TILE_DIM - 1) / TILE_DIM));
  dim3 block_dim(TILE_DIM, TILE_DIM);
  ker_ln_bw_dgamma_dbetta<float><<<grid_dim, block_dim, 0, stream_1>>>(
      d_gamma_grad, d_betta_grad, d_out_grad, d_inp, d_gamma, d_betta, d_vars,
      d_means, batch_size, hidden_dim);

  // Compute grad of input
  if (hidden_dim % 4 != 0 || hidden_dim > 4096) {
    throw std::runtime_error("hidden_dim % 4 != 0 || hidden_dim > 4096");
  }
  hidden_dim >>= 2;
  int nthread = min(((hidden_dim + 31) / 32) * 32, MAX_THREADS);
  ker_ln_bw_dinp<<<batch_size, nthread, 0, stream_2>>>(
      d_inp_grad, d_out_grad, d_inp, d_gamma, d_betta, d_vars, d_means, hidden_dim);

  // Synchronize and check for errors
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "launch_layernorm_bw Error: %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Copy back to host
  hipMemcpy(gamma_grad, d_gamma_grad, gamma_betta_size, hipMemcpyDeviceToHost);
  hipMemcpy(betta_grad, d_betta_grad, gamma_betta_size, hipMemcpyDeviceToHost);
  hipMemcpy(inp_grad, d_inp_grad, grad_output_size, hipMemcpyDeviceToHost);

  // Free device memory
  hipFree(d_gamma_grad);
  hipFree(d_betta_grad);
  hipFree(d_inp_grad);
  hipFree((void *)d_out_grad);
  hipFree((void *)d_inp);
  hipFree((void *)d_gamma);
  hipFree((void *)d_betta);
  hipFree((void *)d_vars);
  hipFree((void *)d_means);
}}
}}
